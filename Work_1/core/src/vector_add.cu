#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../includes/vector_add.cuh"

__global__ void vector_add_kernel(const float* a, const float* b, float* c, int n) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

void vector_add_launch(const float* a, const float* b, float* c, int n) {
  float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
  size_t size = n * sizeof(float);

  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  int blockSize = 256;
  int gridSize = (n + blockSize - 1) / blockSize;

  vector_add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}