#include <cassert>
#include <stdexcept>
#include "../includes/data.cuh"

Data::Data(size_t r, size_t c)
    : rows_(r)
    , cols_(c) {
  const size_t num_bytes = rows_ * cols_ * sizeof(float);
  hipError_t err = hipMalloc(&d_data, num_bytes);
  if (err != hipSuccess) {
    throw std::bad_alloc{};
  }
}

Data::~Data() {
  if (d_data) {
    hipFree(d_data);
  }
}

void Data::fill(const std::vector<float>& host_data) {
  if (host_data.size() != rows_ * cols_) {
    throw std::invalid_argument("host_data size does not match matrix dimensions");
  }
  hipError_t err = hipMemcpy(
      d_data, host_data.data(), rows_ * cols_ * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    throw std::runtime_error("hipMemcpy failed in fill");
  }
}

std::vector<float> Data::to_host() const {
  std::vector<float> host(rows_ * cols_);
  hipError_t err = hipMemcpy(
      host.data(), d_data, rows_ * cols_ * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    throw std::runtime_error("hipMemcpy failed in to_host");
  }
  return host;
}